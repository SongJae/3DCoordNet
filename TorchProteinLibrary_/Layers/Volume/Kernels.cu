#include "hip/hip_runtime.h"
#include </workspace/Molecule3D/TorchProteinLibrary_1/Layers/Volume/Kernels.h>

template <typename T>
__global__ void projectToTensor(T* coords, int* num_atoms_of_type, int* offsets, T *volume, 
                                int spatial_dim, float res){
/*
Input:
        coords: coordinates in a flat array:
                coords: {protein1, ... proteinN}
                protein1: {atom_type1 .. atom_typeM}
                atom_type: {x1,y1,z1 .. xL,yL,zL}
        num_atoms_of_type: number of atoms in each atom_type 
        offsets: offset for coordinates for each atom_type volume
Output: 
	volume: density
*/
	int d = 2;
	int type_index = threadIdx.x;
	T *type_volume = volume + type_index * spatial_dim*spatial_dim*spatial_dim;
	T *atoms_coords = coords + 3*offsets[type_index];
	int n_atoms = num_atoms_of_type[type_index];
	for(int atom_idx = 0; atom_idx<3*n_atoms; atom_idx+=3){
		T 	x = atoms_coords[atom_idx],
			y = atoms_coords[atom_idx + 1],
			z = atoms_coords[atom_idx + 2];
		int x_i = floor(x/res);
		int y_i = floor(y/res);
		int z_i = floor(z/res);
		for(int i=x_i-d; i<=(x_i+d);i++){
			for(int j=y_i-d; j<=(y_i+d);j++){
				for(int k=z_i-d; k<=(z_i+d);k++){
					if( (i>=0 && i<spatial_dim) && (j>=0 && j<spatial_dim) && (k>=0 && k<spatial_dim) ){
						int idx = k + j*spatial_dim + i*spatial_dim*spatial_dim;							
						T r2 = (x - i*res)*(x - i*res)+\
						(y - j*res)*(y - j*res)+\
						(z - k*res)*(z - k*res);
						type_volume[idx]+=exp(-r2/2.0);
					}
				}
			}
		}
	}
}
template <typename T>
__global__ void projectFromTensor(T* coords, T* grad, int* num_atoms_of_type, int* offsets, T *volume,
                                  int spatial_dim, float res){
/*
Input:
	coords: coordinates in a flat array:
		coords: {protein1, ... proteinN}
		protein1: {atom_type1 .. atom_typeM}
		atom_type: {x1,y1,z1 .. xL,yL,zL}
	num_atoms_of_type: number of atoms in each atom_type 
	offsets: offset for coordinates for each atom_type volume
	volume: gradient to be projected on atoms
Output: 
	grad: for each atom to store the gradient projection
*/
	int d = 2;
	int type_index = threadIdx.x;
	T *type_volume = volume + type_index * spatial_dim*spatial_dim*spatial_dim;
	T *atoms_coords = coords + 3*offsets[type_index];
	T *grad_coords = grad + 3*offsets[type_index];
	int n_atoms = num_atoms_of_type[type_index];
	for(int atom_idx = 0; atom_idx<3*n_atoms; atom_idx+=3){
		T 	x = atoms_coords[atom_idx],
			y = atoms_coords[atom_idx + 1],
			z = atoms_coords[atom_idx + 2];
		// grad_coords[atom_idx] = 0.0;
		// grad_coords[atom_idx+1] = 0.0;
		// grad_coords[atom_idx+2] = 0.0;
		int x_i = floor(x/res);
		int y_i = floor(y/res);
		int z_i = floor(z/res);
		
		for(int i=x_i-d; i<=(x_i+d);i++){
			for(int j=y_i-d; j<=(y_i+d);j++){
				for(int k=z_i-d; k<=(z_i+d);k++){
					if( (i>=0 && i<spatial_dim) && (j>=0 && j<spatial_dim) && (k>=0 && k<spatial_dim) ){
						int idx = k + j*spatial_dim + i*spatial_dim*spatial_dim;
						T r2 = (x - i*res)*(x - i*res)+\
						(y - j*res)*(y - j*res)+\
						(z - k*res)*(z - k*res);
						grad_coords[atom_idx] -= (x - i*res)*type_volume[idx]*exp(-r2/2.0);
						grad_coords[atom_idx + 1] -= (y-j*res)*type_volume[idx]*exp(-r2/2.0);
						grad_coords[atom_idx + 2] -= (z-k*res)*type_volume[idx]*exp(-r2/2.0);
					}
				}
			}
		}
		
	}
}


__global__ void selectFromTensor(float *features, 
								float* volume, int spatial_dim, 
								float *coords, int num_atoms, int max_num_atoms,
								float res){
/*
Input:
	volume: 3d array from which we select
	coords: coordinates in a flat array
	num_atoms: number of atoms 
	spatial_dim: volume 3d array real size
	res: volume 3d array resolution
		
Output: 
	features: for each atom to store the elements from the array
*/
	int feature_index = threadIdx.x;
	float *feature_volume = volume + feature_index * spatial_dim*spatial_dim*spatial_dim;
	float *feature_output = features + feature_index * max_num_atoms;
	for(int atom_idx = 0; atom_idx<num_atoms; atom_idx++){
		float 	x = floor(coords[3*atom_idx]/res),
				y = floor(coords[3*atom_idx + 1]/res),
				z = floor(coords[3*atom_idx + 2]/res);
		if( (x<spatial_dim && x>=0)&&(y<spatial_dim && y>=0)&&(z<spatial_dim && z>=0)){
			uint idx = z + y*spatial_dim + x*spatial_dim*spatial_dim;
			feature_output[atom_idx] = feature_volume[idx];
		}
	}
}

template <typename T>
void gpu_computeCoords2Volume(	T *coords,
                                int *num_atoms_of_type,
							    int *offsets, 
								T *volume,
								int spatial_dim,
                                int num_atom_types,
								float res){

	projectToTensor<T><<<1, num_atom_types>>>(	coords, num_atoms_of_type, offsets,
											volume, spatial_dim, res);

}
template <typename T>
void gpu_computeVolume2Coords(	T *coords,
								T* grad,
                                int *num_atoms_of_type,
							    int *offsets, 
								T *volume,
								int spatial_dim,
                                int num_atom_types,
								float res){

	projectFromTensor<T><<<1, num_atom_types>>>(coords, grad, num_atoms_of_type, offsets,
												volume, spatial_dim, res);

}

void gpu_selectFromTensor(	float *features, int num_features, 
							float* volume, int spatial_dim, 
							float *coords, int num_atoms, int max_num_atoms, 
							float res){

	selectFromTensor<<<1, num_features>>>(	features, 
											volume, spatial_dim, 
											coords, num_atoms, max_num_atoms,
											res);
}

template void gpu_computeVolume2Coords<float>(	float*, float*, int*, int*, float*, int, int, float);
template void gpu_computeVolume2Coords<double>(	double*, double*, int*, int*, double*, int, int, float);

template void gpu_computeCoords2Volume<float>(float*, int*, int*, float*, int, int, float);
template void gpu_computeCoords2Volume<double>(double*, int*, int*, double*, int, int, float);